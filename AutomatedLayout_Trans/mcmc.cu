
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <limits.h>
#include <vector>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
using namespace std;
// #define RNG_DEF int& rx
// #define RNG_ARGS rx
// #define MY_RAND_MAX ((1U << 31) - 1)
// Command line arguments that get set below (these give default values)

float sdscale = 1.0; // scale the SDs by this
int N = 1<<20;
int steps = 1000;
int outer = 100;
int thin = 1; // how many outer blocks to skip?
int seed = -1;
int burn = 0;
int QUIET = 0;
int WHICH_GPU = 0;
int FIRST_HALF_DATA = 0;
int EVEN_HALF_DATA = 0;
int SHOW_CONSTANTS = 0;
string in_file_path = "data.txt";
string out_path = "out/";

const unsigned int nBlocks = 10 ;

const unsigned int BLOCK_SIZE = 64;
// const unsigned int HARDARE_MAX_X_BLOCKS = 4096;
// const unsigned int HARDWARE_MAX_THREADS_PER_BLOCK = 1024;
// __device__ __host__ int cuda_rand(RNG_DEF) {
//    //http://rosettacode.org/wiki/Linear_congruential_generator#C
//    return rx = (rx * 1103515245 + 12345) & MY_RAND_MAX;
// }
// __device__ int random_int(int n) {
//      // number in [0,(n-1)]
//     int divisor = MY_RAND_MAX/(n+1);
//     int retval;
//
//     do {
//         retval = cuda_rand(RNG_ARGS) / divisor;
//     } while (retval >= n);
//
//     return retval;
// }
__device__ float density_function(float beta, float cost) {
    // printf("%f-%f\n", beta, cost);
	return exp2f(-beta * cost);
}
__device__ float get_randomNum(unsigned int seed, int maxLimit) {
  /* CUDA's random number library uses curandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t state;

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
              0, /* the sequence number is only important with multiple cores */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);

  /* curand works like rand - except that it takes a state as a parameter */
  return hiprand(&state) % maxLimit;
 // int res = curand(&state) % maxLimit;
 // printf("%d ", res);
 // return res;
}
void setUpDevices(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if(WHICH_GPU <= deviceCount) {
    hipError_t err = hipSetDevice(WHICH_GPU);
    if(err != hipSuccess)
        cout<< "CUDA error:" <<hipGetErrorString(err)<<endl;
    }
    else {
        cout << "Invalid GPU device " << WHICH_GPU << endl;
        exit(-1);
    }
    int wgpu;
    hipGetDevice(&wgpu);
    hipDeviceReset();
}
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i+=stride)
    y[i] = x[i] + y[i];
}
void debug_add(){
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
      x[i] = 1.0f;
      y[i] = 2.0f;
    }

    int numOfBlocks = (N+BLOCK_SIZE-1)/BLOCK_SIZE;
    // Run kernel on 1M elements on the GPU
    add<<<numOfBlocks, BLOCK_SIZE>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
      maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
}
__device__ float cost_function(float * data, int length){
    //dummy cost, just sum up all
    float res = 0;

    for(int i=0; i<length; i++)
        res += data[i];
    // printf("res: %f\n", res);
    return res/1000;
}
__device__
void changeTemparature(float * temparature, unsigned int seed){
    int t1 = get_randomNum(seed, nBlocks);
    int t2=t1;
    while(t2 == t1)
        t2 = get_randomNum(seed + 100, nBlocks);
    float tmp = temparature[t1];
    temparature[t1] = temparature[t2];
    temparature[t2] = tmp;
}
__device__
void ActualHW(int randTimes, int numofObjs, unsigned int seed, int* pickedIdAddr, float*sArray, float * cost, float *temparature){
    // bool hit = false;
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    for(int t=0; t<randTimes; t++){
        if(pickedIdAddr[t] == threadIdx.x){
            if(t % 10 == 0)
                changeTemparature(temparature, seed+index);
            float cost_pri = cost_function(sArray, numofObjs);
            float p0 = density_function(temparature[blockIdx.x], cost_pri);
            float tmpKeep = sArray[threadIdx.x];
            sArray[threadIdx.x] = get_randomNum(seed+index, 1000);

            float cost_post = cost_function(sArray, numofObjs);
            float p = density_function(temparature[blockIdx.x], cost_post);
            float alpha = min(1.0f, p/p0);
            // printf("p/p0: %f\n", p/p0);
            float t =0.8f;
            //change back
            if(alpha>t)
                sArray[threadIdx.x] = tmpKeep;
            else{
                if(sArray[threadIdx.x]>tmpKeep)
                    printf("%f - %f\n", tmpKeep, sArray[threadIdx.x]);
                cost[blockIdx.x] = cost_post;
            }


            // hit = true;
        }
    }
    // return hit;
}
__global__
void simpleHW(int numofObjs, float * gValues, float* gArray,unsigned int seed,int*pickedIdxs, int randTimes){
    //here should be dynamic shared mem
    //__shared__ float sArray[30];
    extern __shared__ float sharedMem[];
    float * sArray = sharedMem;
    float * lastSumUp = (float *) & sArray[nBlocks*numofObjs];
    float * temparature = (float *) & lastSumUp[nBlocks];
    //initialize
    int startIdx = blockIdx.x * numofObjs;
    int idx =  startIdx+ threadIdx.x;

    sArray[idx] = gValues[threadIdx.x];
    temparature[blockIdx.x] = -get_randomNum(seed+blockIdx.x, 100) / 10;
    // printf("temp: %f", temparature[blockIdx.x]);
    lastSumUp[blockIdx.x] = 0;
    for(int i = 0;i<numofObjs; i++)
        lastSumUp[blockIdx.x] += gValues[i];

    int* pickedIdAddr = &pickedIdxs[blockIdx.x * randTimes];

    ActualHW(randTimes, numofObjs, seed, pickedIdAddr, &sArray[startIdx], lastSumUp, temparature);
    __syncthreads();
    gArray[idx] = sArray[idx];
}

void simpleStructure(){
    float *gValues;
    float * gArray;
    int * pickedIdxs;

    int numofObjs = 5;

    int nTimes =20000;

    int totalSize = nBlocks*numofObjs* sizeof(float);

    hipMallocManaged(&gValues, numofObjs * sizeof(float));
    for(int i=0; i<numofObjs; i++)
        gValues[i] = 1000;
    hipMallocManaged(&pickedIdxs, nBlocks*nTimes * sizeof(int));
    for(int i=0; i<nBlocks*nTimes; i++)
        pickedIdxs[i] = rand()%numofObjs;
    // for(int i=0; i<nBlocks*nTimes; i++)
    //     cout<<pickedIdxs[i]<<" ";
    // cout<<endl;

    hipMallocManaged(&gArray, totalSize);
    //dynamic shared mem, <<<nb, nt, sm>>>
    simpleHW<<<nBlocks, numofObjs, totalSize + 2*nBlocks*sizeof(float)>>>(numofObjs, gValues, gArray,time(NULL),pickedIdxs,nTimes);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    for(int i=0;i<nBlocks;i++){
        for(int j=0; j<numofObjs; j++)
            cout<<gArray[i * numofObjs+ j]<<" ";
        cout<<endl;
    }

    // Free memory
    hipFree(gValues);
    hipFree(gArray);
    hipFree(pickedIdxs);
}
int main(int argc, char** argv){
    setUpDevices();
    // debug_add();
    //setup blockSize
    // int N_BLOCKS = N/BLOCK_SIZE + (N%BLOCK_SIZE == 0? 0:1);
    // assert(N_BLOCKS < HARDARE_MAX_X_BLOCKS); // can have at most this many blocks
    // assert(N/N_BLOCKS <= HARDWARE_MAX_THREADS_PER_BLOCK); // MUST HAVE LESS THREADS PER BLOCK!!
    // setup the output files??
    //Make RNG replicable
    if(seed == -1)
        seed = time(NULL);
    srand(seed);
    simpleStructure();
    // read data and se
    return 0;
}
